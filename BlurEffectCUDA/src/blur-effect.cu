#include "hip/hip_runtime.h"
// !./bin/blur-effect $INPUT_FILE $OUTPUT_FILE $KERNEL_SIZE $SIGMA $VERBOSE

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#define STB_IMAGE_IMPLEMENTATION
#include "../lib/stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../lib/stb/stb_image_write.h"
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define SIGMA 15
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* a, double b) { return b; }
#endif

__global__ void calculatePi(double *piTotal, int totalThreads)
{   
    long int i = 0;
    int index = (blockDim.x * blockIdx.x) + threadIdx.x;
     
    __syncthreads();
    if(index == 0){
        for(i = 1; i < totalThreads; i++)
            piTotal[0] += 1;
    }
}

// http://pages.stat.wisc.edu/~mchung/teaching/MIA/reading/diffusion.gaussian.kernel.pdf.pdf
__global__ void generateGaussianKernel(double *k, double *accumulation, int size, double sigma) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int x = idx % size, y = idx / size;
    int i = 0;

    if(idx < size * size) {
        k[idx] = (double)(exp(-(x * x + y * y) / (2.0 * sigma * sigma)) / (2.0 * sigma * sigma * M_PI));
        atomicAdd(accumulation, k[idx]);
    }

    __syncthreads();

    if(idx == 0) {
      for (i = 0; i < (size * size); i++)
        k[i] /= *accumulation;
    }    
}

__device__ void calculatePixel(unsigned char *in, unsigned char *out, long int i, int w, int h, int channels, double* kernel, int kernel_size) {
    int kernel_pad = kernel_size / 2, idx;
    double v = 0.0, total = 0.0;

    for (int l = 0; l < channels; l++) {
        total = 0.0;
        for (int m = -kernel_pad; m <= kernel_pad; m++)
            for (int n = -kernel_pad; n <= kernel_pad; n++) {
                v = kernel[(m  + kernel_pad) * kernel_size + (n + kernel_pad)];
                idx = ((i + l) + (m * w * channels) + (n * channels)) % (w * h * channels);
                total += v * (in[idx]);
            }
        out[i + l] = total;
    }
}

__global__ void applyFilter(unsigned char *in, unsigned char *out, double *kernel, int w, int h, int c, int kernel_size) {
    int kernel_pad = kernel_size / 2;
    size_t size = w * h * c;
    long int idx = (blockDim.x * blockIdx.x + threadIdx.x) * c;

    if(idx < size)
        if(idx >= kernel_pad * w * c && // Top
            idx < (size - kernel_pad * w * c) && // Bottom
            idx % (w * c) >= kernel_pad * c && // Left
            idx % (w * c) < (w * c - kernel_pad * c)) // Right
            calculatePixel(in, out, idx, w, h, c, kernel, kernel_size);
        else 
            for (int j = 0; j < c; j++) 
                out[idx + j] = 0;
}

int main(int argc, char *argv[]) {
    if(argc < 5) {
        printf("Wrong arguments!\n");
        return -1;
    }

    struct timeval after, before, result;
    gettimeofday(&before, NULL);

    char *DIR_IMG_INPUT, *DIR_IMG_OUTPUT;
    int KERNEL_SIZE, THREADS, verbose;
    double sigma;

    DIR_IMG_INPUT = argv[1];
    DIR_IMG_OUTPUT = argv[2];

    KERNEL_SIZE = 3;
    sscanf(argv[3], "%d", &KERNEL_SIZE);
    if(KERNEL_SIZE % 2 == 0) {
        printf("The kernel size must be odd");
        return -1;
    }

    sigma = SIGMA;
    if(argv[4] != 0) sscanf(argv[4], "%lf", &sigma);

    if(argv[5] != NULL) {
        sscanf(argv[5], "%d", &verbose);
        if(verbose != 1)
            verbose = 0;
    }

    int deviceCount = 0;
    CHECK(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA\n");
        return -1;
    }

    if(verbose) printf("Detected %d CUDA Capable device(s)\n", deviceCount);

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int coresPerMP = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
    int multiProcessors = deviceProp.multiProcessorCount;

    if(verbose)
      printf("%d Multiprocessors, %d CUDA Cores/MP | %d CUDA Cores\nMaximum number of threads per block: %d\n",
             deviceProp.multiProcessorCount,
             _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
             _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount,
             deviceProp.maxThreadsPerBlock);
    
    int blocksPerGrid, threadsPerBlock;
    int kernel_cells = KERNEL_SIZE * KERNEL_SIZE;

    double h_kernel[KERNEL_SIZE][KERNEL_SIZE];
    double *d_kernel;

    CHECK(hipMalloc((void **) &d_kernel, kernel_cells * sizeof(double)));

    double *d_sum;
    CHECK(hipMalloc((void **) &d_sum, sizeof(double)));

    threadsPerBlock = MIN(coresPerMP, kernel_cells);
    blocksPerGrid = floor(kernel_cells / threadsPerBlock) + 1;

    generateGaussianKernel<<<blocksPerGrid, threadsPerBlock>>>((double *) d_kernel, d_sum, KERNEL_SIZE, sigma);

    CHECK(hipMemcpy(h_kernel, d_kernel, kernel_cells * sizeof(double), hipMemcpyDeviceToHost));

    if(verbose)
      printf("Kernel computed in %d threads in %d blocks\n.", threadsPerBlock, blocksPerGrid);

    hipFree(d_sum);

    unsigned char *h_data;
    int width, height, channels;
    h_data = stbi_load(DIR_IMG_INPUT, &width, &height, &channels, STBI_default);

    if (h_data != NULL) {
        if(verbose) printf("\nImage dimensions: (%dpx, %dpx) and %d channels.\n", width, height, channels);

        unsigned char *h_output_image, *d_data, *d_output_image;
        CHECK(hipMalloc((void **) &d_output_image, width * height * channels * sizeof(unsigned char)));
        CHECK(hipMalloc((void **) &d_data, width * height * channels * sizeof(unsigned char)));
        CHECK(hipMemcpy(d_data, h_data, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice));
        
        h_output_image = (unsigned char*) malloc(width * height * channels * sizeof(unsigned char));
        if(h_output_image == NULL) {
            printf("Error trying to allocate memory space");
            free(h_output_image);
            stbi_image_free(h_data);
            return -1;
        }

        threadsPerBlock = MIN(coresPerMP * 2, width * height);
        blocksPerGrid = floor(width * height / threadsPerBlock) + 1;

        applyFilter<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_output_image, d_kernel, width, height, channels, KERNEL_SIZE);

        if(verbose)
          printf("Filter applied with %d threads in %d blocks\n.", threadsPerBlock, blocksPerGrid);

        CHECK(hipMemcpy(h_output_image, d_output_image, width * height * channels * sizeof(unsigned char), hipMemcpyDeviceToHost));

        CHECK(hipFree(d_kernel));
        CHECK(hipFree(d_output_image));
        CHECK(hipFree(d_data));

        if (!stbi_write_png(DIR_IMG_OUTPUT, width, height, channels, h_output_image, width * channels))
            printf("Image cannot be created");
        else
            if(verbose) printf("Image created");

        free(h_output_image);
        free(h_data);
    } else {
        printf("Error loading the image");
    }

    gettimeofday(&after, NULL);
    timersub(&after, &before, &result);

    if(verbose) printf("\nTime elapsed: %ld.%06ld\n", (long int) result.tv_sec, (long int) result.tv_usec);
    else
        printf("%ld.%06ld\n", (long int)result.tv_sec, (long int)result.tv_usec);

    stbi_image_free(h_data);
}